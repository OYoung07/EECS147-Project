#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "support.h"

void initVector(float **vec_h, unsigned size)
{
    *vec_h = (float*)malloc(size*sizeof(float));

    if(*vec_h == NULL) {
        FATAL("Unable to allocate host");
    }
    srand(217);
    for (unsigned int i=0; i < size; i++) {
        (*vec_h)[i] = (rand()%100)/100.00;
    }

}


void verify(float* input, unsigned num_elements, float result) {

  const float relativeTolerance = 2e-5;

  float sum = 0.0f;
  for(int i = 0; i < num_elements; ++i) {
    sum += input[i];
  }
  printf("\n Sum: %f \n", sum);
  float relativeError = (sum - result)/sum;
  if (relativeError > relativeTolerance
    || relativeError < -relativeTolerance) {
    printf("TEST FAILED, cpu = %0.3f, gpu = %0.3f\n\n", sum, result);
    exit(0);
  }
  printf("TEST PASSED\n\n");

}

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

unsigned long timerPrompt() {
    int userInput;
    unsigned long long timeLimit;

    printf("Enter max ticks for runtime OR Enter 0 for unlimited ticks: "); //Prompts user to enter maximum ticks
    scanf("%d", &userInput);

    if ("%d", userInput == 0) {
        timeLimit = 2000000000; //basically infinite integer, change type if needed
    }
    else {
        timeLimit = userInput; //user chosen value
    }

    return timeLimit; 
}

float tickTime() {
    float userInput;

    printf("Enter number of secs/tick [RECOMMENDED: 1] : ");
    scanf("%f", &userInput);

    return userInput;
}

unsigned int ticksPerDisplay() {
    unsigned long userInput;

    printf("Enter number of ticks per display frame [RECOMMENDED: 10] : ");
    scanf("%d", &userInput);
    
    return userInput;
}

void writeCSV(unsigned int dataSize, double *data, FILE *fp) {
    if (dataSize == 0) return;      //if 0, outputs nothing
    
    fprintf(fp, "%lf", data[0]);     //output first value
    for (int i = 1; i < dataSize; ++i)
        fprintf(fp, ", %lf", data[i]);   //add value and comma
    fprintf(fp, "\n");
}

