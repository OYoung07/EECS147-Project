#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include "support.h"

void initVector(float **vec_h, unsigned size)
{
    *vec_h = (float*)malloc(size*sizeof(float));

    if(*vec_h == NULL) {
        FATAL("Unable to allocate host");
    }
    srand(217);
    for (unsigned int i=0; i < size; i++) {
        (*vec_h)[i] = (rand()%100)/100.00;
    }

}


void verify(float* input, unsigned num_elements, float result) {

  const float relativeTolerance = 2e-5;

  float sum = 0.0f;
  for(int i = 0; i < num_elements; ++i) {
    sum += input[i];
  }
  printf("\n Sum: %f \n", sum);
  float relativeError = (sum - result)/sum;
  if (relativeError > relativeTolerance
    || relativeError < -relativeTolerance) {
    printf("TEST FAILED, cpu = %0.3f, gpu = %0.3f\n\n", sum, result);
    exit(0);
  }
  printf("TEST PASSED\n\n");

}

void startTime(Timer* timer) {
    gettimeofday(&(timer->startTime), NULL);
}

void stopTime(Timer* timer) {
    gettimeofday(&(timer->endTime), NULL);
}

float elapsedTime(Timer timer) {
    return ((float) ((timer.endTime.tv_sec - timer.startTime.tv_sec) \
                + (timer.endTime.tv_usec - timer.startTime.tv_usec)/1.0e6));
}

unsigned long timerPrompt() {
    int userInput;
    unsigned long long timeLimit;

    printf("Enter max ticks for runtime OR Enter 0 for unlimited ticks: "); //Prompts user to enter maximum ticks
    scanf("%d", &userInput);

    if ("%d", userInput == 0) {
        timeLimit = 2000000000; //basically infinite integer, change type if needed
    }
    else {
        timeLimit = userInput; //user chosen value
    }

    return timeLimit; 
}

float tickTime() {
    float userInput;

    printf("Enter number of secs/tick [DEFAULT: 1] : ");
    scanf("%f", &userInput);

    return userInput;
}

unsigned int ticksPerDisplay() {
    unsigned long userInput;

    printf("Enter number of ticks per display frame [DEFAULT: 100] : ");
    scanf("%d", &userInput);
    
    return userInput;
}
