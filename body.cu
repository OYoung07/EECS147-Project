#include "hip/hip_runtime.h"
#include "body.h"
#include <math.h>
#include <stdio.h>

#define MAX_BODIES 256

/* operator overloading for float3 */
__device__ __host__  float3 operator+(const float3 &a, const float3 &b) {
    float3 c;

    c.x = a.x + b.x; 
    c.y = a.y + b.y; 
    c.z = a.z + b.z;

    return c;
}

__device__  __host__ float3 operator-(const float3 &a, const float3 &b) {
    float3 c;

    c.x = a.x - b.x;
    c.y = a.y - b.y;
    c.z = a.z - b.z;

    return c;
}

__device__  __host__ float3 operator*(const float3 &a, const float &b) {
    float3 c;

    c.x = a.x * b;
    c.y = a.y * b;
    c.z = a.z * b;

    return c;
}

__device__  __host__ float3 operator/(const float3 &a, const float &b) {
    float3 c;
    
    c.x = a.x / b;
    c.y = a.y / b;
    c.z = a.z / b;

    return c;
}

void print_float3(const float3 &f) {
    printf("(%e,%e,%e)", f.x, f.y, f.z);
}

void print_body(struct body* b) {
    printf("Body ID:%d\n",b->id);
    printf("Mass:%e\n",b->mass);
    printf("Radius:%e\n",b->radius);
    printf("Position:");
    print_float3(b->position);
    printf("\nVelocity:");
    print_float3(b->velocity);
    printf("\n");
}

//get distance between two bodies
float distance(struct body* b1, struct body* b2) {
    return sqrt(pow(b2->position.x - b1->position.x, 2) + 
                pow(b2->position.y - b1->position.y, 2) + 
                pow(b2->position.z - b1->position.z, 2));
}

//get gravity force magnitude between two bodies
float calculate_FG(struct body* b1, struct body* b2) {
    double G = 6.674e-11;
    double d = distance(b1, b2);
    double mag_F; 

    mag_F = (G * (double)b1->mass *(double)b2->mass)/pow(d, 2); //gravity formula

    return (float)mag_F;
}

//get direction vector between two bodies
float3 get_direction_vector(struct body* origin, struct body* actor) {
    float3 direction;
    float norm = distance(origin, actor);

    direction = actor->position - origin->position;
    direction = direction / norm;

    return direction;
}

/* calculate acceleration of origin as exerted by actor */
float3 get_accel_vector(struct body* origin, struct body* actor) {
    float F = calculate_FG(origin, actor);
    float3 dir = get_direction_vector(origin, actor);

    float3 F_vec = dir * F; //get force vector
    float3 A_vec = F_vec / origin->mass; //F = MA -> A = F/M

    return A_vec;
}

//calculate mean acceleration vector from all other bodies
float3 CPU_reduce_accel_vectors(struct body b, struct body* bodies, const int &num_bodies) {
    float3 accel;
    accel.x = 0;
    accel.y = 0;
    accel.z = 0;    

    for (int i = 0; i < num_bodies; i++) {
        if (bodies[i].id != b.id) { //if not self
           accel = accel + get_accel_vector(&b, &bodies[i]); 
        }
    }

    return accel;
}

//collide two bodies inelasticly and return the resultant body
__device__ __host__ struct body create_new_body(struct body* a, struct body* b) {
    struct body c;

    c.mass = a->mass + b->mass;
    c.radius = cbrt((pow(a->radius,3) +  pow(b->radius,3))); //combine radii and conserve area

    if (a->mass >= b->mass) {
        c.position = a->position;
    } else {
        c.position = b->position;
    }

    c.velocity = ((a->velocity * a->mass) + (b->velocity * b->mass))/c.mass;

    return c;
}

__device__ __host__ unsigned int delete_body_id(unsigned int id, struct body* bodies, const int &num_bodies) {
    unsigned int delete_index = num_bodies;

    for (int i = 0; i < num_bodies; i++) {
        if (bodies[i].id = id) {
            delete_index = i;
            break;
        }
    }

    if (delete_index < (num_bodies - 1)) {
        for (int i = delete_index; i < (num_bodies - 1); i++) {
            bodies[i] = bodies[i+1];
        }
        return num_bodies - 1;
    } else if (delete_index == (num_bodies - 1)) {
        return num_bodies - 1;
    }

    return num_bodies;    
}

unsigned int CPU_collisions(struct body* bodies, int num_bodies) {
    unsigned int marked_for_death[MAX_BODIES];
    unsigned int death_index = 0;

    struct body new_bodies[10];
    unsigned int new_body_index = 0;

    unsigned int temp_id;    

    for (int i = (num_bodies - 1); i >= 0; i--) {
        for(int j = 0; j < num_bodies; j++) {
            for (int k = 0; k < death_index; k++) {
                if (marked_for_death[k] = bodies[i].id) {
                    num_bodies = delete_body_id(bodies[i].id, bodies, num_bodies);
                    break;
                }
            }

            if ((distance(&bodies[i], &bodies[j]) < (bodies[i].radius + bodies[j].radius)) && (i != j)) {
                marked_for_death[death_index] = bodies[j].id;
                death_index++;
                
                new_bodies[new_body_index] = create_new_body(&bodies[i], &bodies[j]);
                new_bodies[new_body_index].id = bodies[i].id; //we know it will be unused
                new_body_index++;
                
                num_bodies = delete_body_id(bodies[i].id, bodies, num_bodies);
                break;              
            }
        }
    }

    for (int i = 0; i < new_body_index; i++) {
        bodies[num_bodies] = new_bodies[i];
        num_bodies++;   
    }

    return num_bodies;
}

void CPU_tick(struct body* bodies, const int &num_bodies, const float &t) {
    float3 a;    

    /* allocate temp array for calculation */
    struct body* temp_bodies;
    temp_bodies = (struct body*) malloc(num_bodies * sizeof(struct body));
    memcpy(temp_bodies, bodies, (num_bodies * sizeof(struct body)));

    for (int i = 0; i < num_bodies; i++) {
        a = CPU_reduce_accel_vectors(bodies[i], temp_bodies, num_bodies);
        
        bodies[i].velocity = bodies[i].velocity + (a * (t/2.0)); //kick        
        bodies[i].position = bodies[i].position + (bodies[i].velocity * t); //drift
       
        a = CPU_reduce_accel_vectors(bodies[i], temp_bodies, num_bodies);

        bodies[i].velocity = bodies[i].velocity + (a * (t/2.0)); //kick 
    }

    free(temp_bodies); //memory leaks are bad
}

void print_bodies(struct body* bodies, const int &num_bodies, const float &tile_scale) {
    char map[40][40];
    float y_index;
    float x_index;    

    //draw true to size
    for (int y = 0; y < 40; y++) {
        for (int x = 0; x < 40; x++) {
            map[y][x] = ' ';
            for (int i = 0; i < num_bodies; i++) {
                if (sqrt(pow(bodies[i].position.x - ((x-20.0) * tile_scale), 2.0) + pow(bodies[i].position.y - ((y-20.0) * tile_scale), 2.0)) < bodies[i].radius) {
                    map[y][x] = '@';
                }  
            }
        }     
    }

    //draw as point mass if too small
    for (int i = 0; i < num_bodies; i++) {
        y_index = (bodies[i].position.y / tile_scale) + 20;
        x_index = (bodies[i].position.x / tile_scale) + 20;
        
        if (y_index < 40 && y_index >= 0 && x_index < 40 && x_index >= 0) {
            if (map[(int)y_index][(int)x_index] != '@') {
                map[(int)y_index][(int)x_index] = '.';
            }
        }
    }

    //print
    printf("\e[1;1H\e[2J"); //clear screen
    for (int y = 0; y < 40; y++) {
        for (int x = 0; x < 40; x++) {
            printf(" %c",map[y][x]);
        }
        printf("\n");
    }
}

void print_bodies_numbered(struct body* bodies, const int &num_bodies, const float &tile_scale) {
    char map[40][40];
    int y_index;
    int x_index;    

    for (int y = 0; y < 40; y++) {
        for (int x = 0; x < 40; x++) {
            map[y][x] = ' ';
        }
    }

    for (int i = 0; i < num_bodies; i++) {
        y_index = (int)(bodies[i].position.y / tile_scale) + 20;
        x_index = (int)(bodies[i].position.x / tile_scale) + 20;
        
        if (y_index < 40 && y_index >= 0 && x_index < 40 && x_index >= 0) {
            map[y_index][x_index] = ((char)bodies[i].id % 90) + 33; //give unique character
        }
    }

    //print
    printf("\e[1;1H\e[2J"); //clear screen
    for (int y = 0; y < 40; y++) {
        for (int x = 0; x < 40; x++) {
            printf(" %c",map[y][x]);
        }
        printf("\n");
    }
}
