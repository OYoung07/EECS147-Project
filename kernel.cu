#include "hip/hip_runtime.h"
#include <stdio.h>
#include "body.h"

#define BLOCK_SIZE 16

__device__ inline float3 operator+(const float3 &a, const float3 &b) {
    float3 c;

    c.x = a.x + b.x; 
    c.y = a.y + b.y; 
    c.z = a.z + b.z;

    return c;
}

__device__ inline float3 operator-(const float3 &a, const float3 &b) {
    float3 c;

    c.x = a.x - b.x;
    c.y = a.y - b.y;
    c.z = a.z - b.z;

    return c;
}

__device__ inline float3 operator*(const float3 &a, const float &b) {
    float3 c;

    c.x = a.x * b;
    c.y = a.y * b;
    c.z = a.z * b;

    return c;
}

__device__ inline float3 operator/(const float3 &a, const float &b) {
    float3 c;
    
    c.x = a.x / b;
    c.y = a.y / b;
    c.z = a.z / b;

    return c;
}


//get distance between two bodies
__device__ float GPU_distance(struct body* b1, struct body* b2) {
    return sqrt(pow(b2->position.x - b1->position.x, 2) + 
                pow(b2->position.y - b1->position.y, 2) + 
                pow(b2->position.z - b1->position.z, 2));
}


//get gravity force magnitude between two bodies
__device__ float GPU_calculate_FG(struct body* b1, struct body* b2) {
    float G = 6.674e-11;
    float d = GPU_distance(b1, b2);
    float mag_F; 

    mag_F = (G * b1->mass * b2->mass)/pow(d, 2); //gravity formula

    return mag_F;
}

//get direction vector between two bodies
__device__ float3 GPU_get_direction_vector(struct body* origin, struct body* actor) {
    float3 direction;
    float norm = GPU_distance(origin, actor);

    direction = actor->position - origin->position;
    direction = direction / norm;

    return direction;
}

/* calculate acceleration of origin as exerted by actor */
__device__ float3 GPU_get_accel_vector(struct body* origin, struct body* actor) {
    float F = GPU_calculate_FG(origin, actor);
    float3 dir = GPU_get_direction_vector(origin, actor);

    float3 F_vec = dir * F; //get force vector
    float3 A_vec = F_vec / origin->mass; //F = MA -> A = F/M

    return A_vec;
}



//GPU kernel
//TODO: need to redefine body functions as host functions
__global__ void GPU_reduce_accel_vectors(float3* accel_out, struct body* b, struct body** bodies, const unsigned  int num_bodies) {
    float3 accel;
    accel.x = 0;
    accel.y = 0;
    accel.z = 0;    
 
    __shared__ float3 partialSum[2 * BLOCK_SIZE];

    unsigned int tx = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    if (bodies[tx]->id == b->id) {
        return; //exit if current body is self
    }

    partialSum[tx] = GPU_get_accel_vector(b, bodies[start + tx]);
    partialSum[blockDim.x + tx] = GPU_get_accel_vector(b, bodies[start + blockDim.x + tx]);

    for (unsigned int stride = blockDim.x; stride > 0; stride /= 2) {
        __syncthreads();

        if (tx < stride) {
            partialSum[tx] = partialSum[tx] +  partialSum[tx + stride];
        }
    }

    if (tx == 0) {
        atomicAdd(&accel_out->x, partialSum[0].x); //add back to output without race condition
        atomicAdd(&accel_out->y, partialSum[0].y);
        atomicAdd(&accel_out->z, partialSum[0].z);
    }
}

float3 GPU_calculate_acceleration(struct body** CPU_bodies, struct body* CPU_b, const unsigned int num_bodies) {
    float3 CPU_accel;
    float3* GPU_accel;
    struct body* GPU_b;
    struct body** GPU_bodies;

    dim3 DimBlock(BLOCK_SIZE);
    dim3 DimGrid(num_bodies/(BLOCK_SIZE * 2));

    hipMalloc((void**) &GPU_accel, sizeof(float3));
    hipMalloc((void**) &GPU_b, sizeof(struct body)); //will be written to
    hipMalloc((void**) &GPU_bodies, sizeof(struct body) * num_bodies); //will be read-only
    
    CPU_accel.x = 0;
    CPU_accel.y = 0;
    CPU_accel.z = 0;
    
    hipMemcpy(&GPU_accel, &CPU_accel, sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(GPU_b, CPU_b, sizeof(struct body), hipMemcpyHostToDevice);
    hipMemcpy(GPU_bodies, CPU_bodies, sizeof(struct body) * num_bodies, hipMemcpyHostToDevice);

    GPU_reduce_accel_vectors<<<DimGrid,DimBlock>>>(GPU_accel, GPU_b, GPU_bodies, num_bodies);

    hipMemcpy(&CPU_accel, GPU_accel, sizeof(float3), hipMemcpyDeviceToHost);
    hipFree(GPU_accel);
    hipFree(GPU_b);
    hipFree(GPU_bodies);
    
    return CPU_accel;
}

//need to allocate GPU memory for bodies and accel_out
/*
void GPU_tick(struct body* CPU_bodies, const int &num_bodies) {
    float3 CPU_a; //acceleration scalar
    float3 GPU_a;
    struct body* GPU_bodies; //deivce memory for computation
    struct body* GPU_body_outputs; //array to write to

    hipMalloc((void**) &GPU_bodies, sizeof(struct body) * num_bodies); //read-only
    hipMalloc((void**) &GPU_body_outputs, sizeof(struct body) * num_bodies); //write to

    hipMemcpy(GPU_bodies, CPU_bodies, sizeof(struct body) * num_bodies, hipMemcpyHostToDevice);
    hipMemcpy(GPU_body_outputs, CPU_bodies, sizeof(struct body) * num_bodiesi, hipMemcpyHostToDevice);

    for (int i = 0; i < num_bodies, i++) {
        //copy to GPU and run acceleration calculations
        CPU_a = 0;
        hipMemcpy(GPU_a, CPU_a, sizeof(float3), hipMemcpyHostToDevice);    

        GPU_reduce_accel_vectors(GPU_a, GPU_body_outputs[i], GPU_bodies, num_bodies);

        hipMemcpy(CPU_a, GPU_a, sizeof(float3), hipMemcpyHostToDevice);
 
        GPU_bo       
    }

    //just for porting, delete this late:
    for (int i = 0; i < num_bodies; i++) {
        a = CPU_reduce_accel_vectors(bodies[i], bodies, num_bodies);
        
        bodies[i]->velocity = bodies[i]->velocity + (a * (t/2.0)); //kick        
        bodies[i]->position = bodies[i]->position + (bodies[i]->velocity * t); //drift
       
        a = CPU_reduce_accel_vectors(bodies[i], bodies, num_bodies);

        bodies[i]->velocity = bodies[i]->velocity + (a * (t/2.0)); //kick 
    }
    hipMemcpy(GPU_bodies, bodies, sizeof(struct body) * num_bodies);
}
*/
