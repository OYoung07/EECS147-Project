#include "hip/hip_runtime.h"
#include <stdio.h>
#include "body.h"

#define BLOCK_SIZE 256
#define SHARED_MEM_SIZE 256

__device__ void float3_atomicAdd(float3* f, float3 addend) {
    atomicAdd(&(f->x), addend.x);
    atomicAdd(&(f->y), addend.y);
    atomicAdd(&(f->z), addend.z);
}

//get distance between two bodies
__device__ float GPU_distance(struct body* b1, struct body* b2) {
    return sqrt(pow(b2->position.x - b1->position.x, 2) + 
                pow(b2->position.y - b1->position.y, 2) + 
                pow(b2->position.z - b1->position.z, 2));
}


//get gravity force magnitude between two bodies
__device__ float GPU_calculate_FG(struct body* b1, struct body* b2) {
    double G = 6.674e-11;
    double d = GPU_distance(b1, b2);
    double mag_F; 

    mag_F = (G *(double)b1->mass *(double)b2->mass)/pow(d, 2); //gravity formula

    return (float)mag_F;
}

//get direction vector between two bodies
__device__ float3 GPU_get_direction_vector(struct body* origin, struct body* actor) {
    float3 direction;
    float norm = GPU_distance(origin, actor);

    direction = actor->position - origin->position;
    direction = direction / norm;

    return direction;
}

/* calculate acceleration of origin as exerted by actor */
__device__ float3 GPU_get_accel_vector(struct body* origin, struct body* actor) {
    float F = GPU_calculate_FG(origin, actor);
    float3 dir = GPU_get_direction_vector(origin, actor);

    float3 F_vec = dir * F; //get force vector
    float3 A_vec = F_vec / origin->mass; //F = MA -> A = F/M

    return A_vec;
}


//GPU kernel
__global__ void GPU_reduce_accel_vectors(float3* accel_out, struct body b, struct body* bodies, const unsigned int num_bodies) {
    float3 body_accel;

    unsigned int tx = threadIdx.x;
    unsigned int bx = blockIdx.x;

    unsigned int index = tx + (bx * BLOCK_SIZE);

    if (index < num_bodies) {
        if (b.id != bodies[index].id) {    
            body_accel = GPU_get_accel_vector(&b, &bodies[index]);
            float3_atomicAdd(accel_out, body_accel);
        }
    }

    __syncthreads();
}

float3 GPU_calculate_acceleration(struct body b, struct body* CPU_bodies, const unsigned int num_bodies) {
    hipError_t cuda_ret;
    float3 CPU_accel;
    float3* GPU_accel;
    struct body* GPU_bodies;

    dim3 DimBlock(BLOCK_SIZE, 1, 1);
    dim3 DimGrid(ceil((float)num_bodies/((float)BLOCK_SIZE)), 1, 1);

    hipMalloc((void**) &GPU_accel, sizeof(float3));
    hipMalloc((void**) &GPU_bodies, sizeof(struct body) * num_bodies); //will be read-only
 
    hipDeviceSynchronize();   
 
    CPU_accel.x = 0;
    CPU_accel.y = 0;
    CPU_accel.z = 0;
    
    hipMemcpy(GPU_accel, &CPU_accel, sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(GPU_bodies, CPU_bodies, sizeof(struct body) * num_bodies, hipMemcpyHostToDevice);

    hipDeviceSynchronize();

    GPU_reduce_accel_vectors<<<DimGrid,DimBlock>>>(GPU_accel, b, GPU_bodies, num_bodies);

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(cuda_ret));
        printf("Oppsie woopsie I did a fucky wucky (GPU kernel failed, lmao)\n");
    }

    hipMemcpy(&CPU_accel, GPU_accel, sizeof(float3), hipMemcpyDeviceToHost);
 
    hipDeviceSynchronize();

    hipFree(GPU_accel);
    hipFree(GPU_bodies);
    
    hipDeviceSynchronize();   

    return CPU_accel;
}

void GPU_tick(struct body* bodies, const int &num_bodies, const float &t) {
    float3 a;    

    for (int i = 0; i < num_bodies; i++) {
        a = GPU_calculate_acceleration(bodies[i], bodies, num_bodies);
        
        bodies[i].velocity = bodies[i].velocity + (a * (t/2.0)); //kick        
        bodies[i].position = bodies[i].position + (bodies[i].velocity * t); //drift
       
        a = GPU_calculate_acceleration(bodies[i], bodies, num_bodies);

        bodies[i].velocity = bodies[i].velocity + (a * (t/2.0)); //kick 
    }
}

//better GPU kernel
__global__ void GPU_tick_shared_memory(struct body* output_bodies, const unsigned int num_bodies, const float t) {
    __shared__ struct body temp_bodies_shared[SHARED_MEM_SIZE]; 

    unsigned int tx = threadIdx.x;
    unsigned int bx = blockIdx.x;

    unsigned int index = tx + (bx * BLOCK_SIZE);

    float3 a;

    if (index < num_bodies) { //populate shared memory
       temp_bodies_shared[index] = output_bodies[index]; 
    }

    __syncthreads();

    //do calculations 
    if (index < num_bodies) {
        //get first acceleration
        a.x = 0; a.y = 0; a.z = 0;
        for (int i = 0; i < num_bodies; i++) { 
            if (output_bodies[index].id != temp_bodies_shared[i].id) {    
                a = a + GPU_get_accel_vector(&output_bodies[index], &temp_bodies_shared[i]);
            }
        }
        
        output_bodies[index].velocity = output_bodies[index].velocity + (a * (t/2.0)); //kick        
        output_bodies[index].position = output_bodies[index].position + (output_bodies[index].velocity * t); //drift
                  
        //get second acceleration
        a.x = 0; a.y = 0; a.z = 0;
        for (int i = 0; i < num_bodies; i++) { 
            if (output_bodies[index].id != temp_bodies_shared[i].id) {    
                a = a + GPU_get_accel_vector(&output_bodies[index], &temp_bodies_shared[i]);
            }
        }
        
        output_bodies[index].velocity = output_bodies[index].velocity + (a * (t/2.0)); //kick 
    }
}

void GPU_tick_improved(struct body* CPU_bodies, const unsigned int &num_bodies, const float &t) {
    hipError_t cuda_ret;
    struct body* GPU_bodies;

    dim3 DimBlock(BLOCK_SIZE, 1, 1);
    dim3 DimGrid(ceil((float)num_bodies/((float)BLOCK_SIZE)), 1, 1);

    hipMalloc((void**) &GPU_bodies, sizeof(struct body) * num_bodies); 
    hipDeviceSynchronize();   
 
    hipMemcpy(GPU_bodies, CPU_bodies, sizeof(struct body) * num_bodies, hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    GPU_tick_shared_memory<<<DimGrid,DimBlock>>>(GPU_bodies, num_bodies, t);

    cuda_ret = hipDeviceSynchronize();
    if (cuda_ret != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(cuda_ret));
        printf("Oppsie woopsie I did a fucky wucky (GPU kernel failed, lmao)\n");
    }

    hipMemcpy(CPU_bodies, GPU_bodies, sizeof(struct body) * num_bodies, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(GPU_bodies);
    hipDeviceSynchronize();
}
